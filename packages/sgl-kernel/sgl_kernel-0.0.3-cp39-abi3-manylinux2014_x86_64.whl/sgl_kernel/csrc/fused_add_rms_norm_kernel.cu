#include "hip/hip_runtime.h"
// Adapted from https://github.com/flashinfer-ai/flashinfer/blob/v0.1.6/include/flashinfer/norm.cuh
// and https://github.com/flashinfer-ai/flashinfer/blob/v0.1.6/python/csrc/norm.cu
// TODO(zhyncs): tmp fix, v0.1.6 enables SGLang e2e to pass CIs unlike v0.2.0

#include <ATen/cuda/HIPContext.h>

#include <flashinfer/math.cuh>
#include <flashinfer/utils.cuh>
#include <flashinfer/vec_dtypes.cuh>
#include <numeric>

#include "utils.h"

using namespace flashinfer;

template <uint32_t VEC_SIZE, typename T>
__global__ void FusedAddRMSNormKernel(T* __restrict__ input, T* __restrict__ residual, T* __restrict__ weight,
                                      const uint32_t d, float eps) {
  const uint32_t bx = blockIdx.x;
  const uint32_t tx = threadIdx.x, ty = threadIdx.y;
  constexpr uint32_t warp_size = 32;
  const uint32_t num_warps = blockDim.y;
  const uint32_t thread_id = tx + ty * warp_size;
  const uint32_t num_threads = num_warps * warp_size;
  const uint32_t rounds = ceil_div(d, VEC_SIZE * num_threads);
  extern __shared__ float smem[];

  float sum_sq = 0.f;

  for (uint32_t i = 0; i < rounds; i++) {
    vec_t<T, VEC_SIZE> input_vec;
    input_vec.fill(0.f);
    vec_t<T, VEC_SIZE> residual_vec;
    residual_vec.fill(0.f);
    if ((i * num_threads + thread_id) * VEC_SIZE < d) {
      input_vec.load(input + bx * d + i * num_threads * VEC_SIZE + thread_id * VEC_SIZE);
      residual_vec.load(residual + bx * d + i * num_threads * VEC_SIZE + thread_id * VEC_SIZE);
    }
#pragma unroll
    for (uint32_t j = 0; j < VEC_SIZE; j++) {
      float x = float(input_vec[j]);
      x += float(residual_vec[j]);
      sum_sq += x * x;
      residual_vec[j] = (T)x;
    }
    if ((i * num_threads + thread_id) * VEC_SIZE < d) {
      residual_vec.store(residual + bx * d + i * num_threads * VEC_SIZE + thread_id * VEC_SIZE);
    }
  }

  // first, warp reduce sum
#pragma unroll
  for (uint32_t offset = warp_size / 2; offset > 0; offset /= 2) {
    sum_sq += math::shfl_xor_sync(sum_sq, offset);
  }

  smem[ty] = sum_sq;
  __syncthreads();
  // then, cross warp reduce sum using only the first warp
  if (ty == 0) {
    sum_sq = (tx < num_warps) ? smem[tx] : 0.f;
#pragma unroll
    for (uint32_t offset = warp_size / 2; offset > 0; offset /= 2) {
      sum_sq += math::shfl_xor_sync(sum_sq, offset);
    }
    smem[0] = sum_sq;
  }
  __syncthreads();

  float rms_rcp = math::rsqrt(smem[0] / float(d) + eps);

  for (uint32_t i = 0; i < rounds; i++) {
    vec_t<T, VEC_SIZE> input_vec;
    vec_t<T, VEC_SIZE> weight_vec;
    vec_t<T, VEC_SIZE> residual_vec;
    input_vec.fill(0.f);
    weight_vec.fill(0.f);
    residual_vec.fill(0.f);
    if ((i * num_threads + thread_id) * VEC_SIZE < d) {
      input_vec.load(input + bx * d + i * num_threads * VEC_SIZE + thread_id * VEC_SIZE);
      weight_vec.load(weight + i * num_threads * VEC_SIZE + thread_id * VEC_SIZE);
      residual_vec.load(residual + bx * d + i * num_threads * VEC_SIZE + thread_id * VEC_SIZE);
    }
#pragma unroll
    for (uint32_t j = 0; j < VEC_SIZE; j++) {
      input_vec[j] = float(residual_vec[j]) * rms_rcp * float(weight_vec[j]);
    }
    if ((i * num_threads + thread_id) * VEC_SIZE < d) {
      input_vec.store(input + bx * d + i * num_threads * VEC_SIZE + thread_id * VEC_SIZE);
    }
  }
}

template <typename T>
hipError_t FusedAddRMSNorm(T* input, T* residual, T* weight, uint32_t batch_size, uint32_t d, float eps = 1e-5,
                            hipStream_t stream = 0) {
  const uint32_t vec_size = std::gcd(16 / sizeof(T), d);

  const uint32_t block_size = std::min<uint32_t>(1024, d / vec_size);
  const uint32_t num_warps = ceil_div(block_size, 32);
  dim3 nblks(batch_size);
  dim3 nthrs(32, num_warps);
  const uint32_t smem_size = num_warps * sizeof(float);
  void* args[] = {&input, &residual, &weight, &d, &eps};

  DISPATCH_ALIGNED_VEC_SIZE(vec_size, VEC_SIZE, {
    auto kernel = FusedAddRMSNormKernel<VEC_SIZE, T>;
    FLASHINFER_CUDA_CALL(hipLaunchKernel((void*)kernel, nblks, nthrs, args, smem_size, stream));
  });

  return hipSuccess;
}

void sgl_fused_add_rmsnorm(torch::Tensor input, torch::Tensor residual, torch::Tensor weight, double eps) {
  CHECK_INPUT(input);
  CHECK_INPUT(residual);
  CHECK_INPUT(weight);
  auto device = input.device();
  CHECK_EQ(residual.device(), device);
  CHECK_EQ(weight.device(), device);
  CHECK_DIM(2, input);     // input: (batch_size, hidden_size)
  CHECK_DIM(2, residual);  // residual: (batch_size, hidden_size)
  CHECK_DIM(1, weight);    // weight: (hidden_size)
  CHECK_EQ(input.size(0), residual.size(0));
  CHECK_EQ(input.size(1), residual.size(1));
  CHECK_EQ(input.size(1), weight.size(0));
  unsigned int batch_size = input.size(0);
  unsigned int hidden_size = input.size(1);

  hipStream_t torch_current_stream = at::cuda::getCurrentCUDAStream();
  // support float16, bfloat16 and float32
  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FLOAT_FP16(input.scalar_type(), c_type, [&] {
    hipError_t status =
        FusedAddRMSNorm(static_cast<c_type*>(input.data_ptr()), static_cast<c_type*>(residual.data_ptr()),
                        static_cast<c_type*>(weight.data_ptr()), batch_size, hidden_size, eps, torch_current_stream);
    TORCH_CHECK(status == hipSuccess,
                "FusedAddRMSNorm failed with error code " + std::string(hipGetErrorString(status)));
    return true;
  });
}
