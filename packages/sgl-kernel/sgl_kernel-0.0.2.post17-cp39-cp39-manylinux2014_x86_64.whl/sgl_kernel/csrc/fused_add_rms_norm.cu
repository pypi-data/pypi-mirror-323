#include "hip/hip_runtime.h"
// Adapted from
// https://github.com/InternLM/lmdeploy/blob/800b6010c0bf76aadf678bc38a507b749fb9774c/src/turbomind/kernels/norm/rms_norm.cu

#include <turbomind/kernels/core/array_ops.h>
#include <turbomind/kernels/core/common.h>

#include <cub/block/block_reduce.cuh>

using namespace turbomind;

template <class T, class Tacc, int block_dim, int vec_size>
__global__ void BiasResidualRMSNormKernel(T* __restrict__ residual, T* __restrict__ hidden_states,
                                          const T* __restrict__ weights, const T* __restrict__ bias, int dims, int num,
                                          float eps, float inv_dims) {
  const int ti = blockIdx.x;
  const int di = threadIdx.x * vec_size;

  if (ti >= num) {
    return;
  }

  residual += dims * ti;
  hidden_states += dims * ti;

  Array<Tacc, vec_size> accum{};

  Array<T, vec_size> r_vec;
  Array<T, vec_size> h_vec;
  Array<T, vec_size> b_vec;

  for (int i = di; i < dims; i += block_dim * vec_size) {
    Load(r_vec, &residual[i]);
    Load(h_vec, &hidden_states[i]);

    using namespace ops;
    r_vec = r_vec + h_vec;

    if (bias) {
      Ldg(b_vec, &bias[i]);
      r_vec = r_vec + b_vec;
    }

    Store(&residual[i], r_vec);

    Array<Tacc, vec_size> tmp = cast<Tacc>(r_vec);

    accum = accum + tmp * tmp;
  }

  float sum{};
  PRAGMA_UNROLL
  for (int i = 0; i < vec_size; ++i) {
    sum += accum[i];
  }

  using BlockReduce = hipcub::BlockReduce<Tacc, block_dim>;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  sum = BlockReduce{temp_storage}.Sum(sum);

  __shared__ float shared_sum;

  if (threadIdx.x == 0) {
    shared_sum = rsqrtf(sum * inv_dims + eps);
  }

  __syncthreads();

  sum = shared_sum;

  Array<T, vec_size> w_vec;
  for (int i = di; i < dims; i += block_dim * vec_size) {
    Load(r_vec, &residual[i]);
    Ldg(w_vec, &weights[i]);
    PRAGMA_UNROLL
    for (int c = 0; c < vec_size; ++c) {
      r_vec[c] = (T)((float)r_vec[c] * sum) * w_vec[c];
    }
    Store(&hidden_states[i], r_vec);
  }
}

template <class T>
void invokeBiasResidualRMSNorm(T* residual, T* hidden_states, const T* weights, const T* bias, int dims, int num,
                               float eps, hipStream_t st) {
  constexpr int vec_size = 16 / sizeof(T);
  constexpr int threads = 512;
  const int blocks = num;

  BiasResidualRMSNormKernel<T, float, threads, vec_size>
      <<<blocks, threads, 0, st>>>(residual, hidden_states, weights, bias, dims, num, eps, 1.f / dims);
}
