#include "hip/hip_runtime.h"
#include <vector>

#include "utils.h"

// trt_reduce
using fptr_t = int64_t;
fptr_t init_custom_ar(int64_t rank_id, int64_t world_size, torch::Tensor& rank_data, const std::vector<fptr_t>& buffers,
                      const std::vector<fptr_t>& tmp_result_buffers, const std::vector<fptr_t>& barrier_in,
                      const std::vector<fptr_t>& barrier_out);
void dispose(fptr_t _fa);
void all_reduce(fptr_t _fa, torch::Tensor& inp, torch::Tensor& out);
std::tuple<std::vector<int64_t>, std::vector<int64_t>> get_graph_buffer_ipc_meta(fptr_t _fa);
void register_graph_buffers(fptr_t _fa, const std::vector<std::vector<int64_t>>& handles,
                            const std::vector<std::vector<int64_t>>& offsets);

// moe_align_block_size
void moe_align_block_size(torch::Tensor topk_ids, int64_t num_experts, int64_t block_size,
                          torch::Tensor sorted_token_ids, torch::Tensor experts_ids, torch::Tensor num_tokens_post_pad,
                          torch::Tensor token_cnts_buffer, torch::Tensor cumsum_buffer);

// sampling_scaling_penalties
torch::Tensor sampling_scaling_penalties(const torch::Tensor& logits, const torch::Tensor& scaling_penalties);

// int8_scaled_mm
torch::Tensor int8_scaled_mm(const torch::Tensor& mat_a, const torch::Tensor& mat_b, const torch::Tensor& scales_a,
                             const torch::Tensor& scales_b, const torch::Dtype& out_dtype,
                             const c10::optional<torch::Tensor>& bias);

// lightning_attention_decode
void lightning_attention_decode(const torch::Tensor& q, const torch::Tensor& k, const torch::Tensor& v,
                                const torch::Tensor& past_kv, const torch::Tensor& slope, torch::Tensor output,
                                torch::Tensor new_kv);

// rotary embedding
void rotary_embedding(torch::Tensor& positions, torch::Tensor& query, torch::Tensor& key, int64_t head_size,
                      torch::Tensor& cos_sin_cache, bool is_neox);

// rms norm
void rmsnorm(at::Tensor& output, at::Tensor& input, at::Tensor& weight, double eps, int64_t hip_stream);

// fused rms norm
void fused_add_rmsnorm(at::Tensor& input, at::Tensor& residual, at::Tensor& weight, double eps, int64_t hip_stream);

// gemma rms norm
void gemma_rmsnorm(at::Tensor& output, at::Tensor& input, at::Tensor& weight, double eps, int64_t hip_stream);

// fused gemma rms norm
void gemma_fused_add_rmsnorm(at::Tensor& input, at::Tensor& residual, at::Tensor& weight, double eps,
                             int64_t hip_stream);

// silu and mul
void silu_and_mul(at::Tensor& out, at::Tensor& input, int64_t hip_stream);

// gelu tanh and mul
void gelu_tanh_and_mul(at::Tensor& out, at::Tensor& input, int64_t hip_stream);

// gelu and mul
void gelu_and_mul(at::Tensor& out, at::Tensor& input, int64_t hip_stream);

// bmm fp8
void bmm_fp8(at::Tensor A, at::Tensor B, at::Tensor D, at::Tensor A_scale, at::Tensor B_scale,
             at::Tensor workspace_buffer, int64_t cublas_handle, int64_t hip_stream);

// min p sampling from probs
void min_p_sampling_from_probs(at::Tensor probs, at::Tensor uniform_samples, at::Tensor samples,
                               std::optional<at::Tensor> maybe_min_p_arr, double min_p_val, bool deterministic,
                               int64_t hip_stream);

// top k renorm probs
void top_k_renorm_probs(at::Tensor probs, at::Tensor renorm_probs, std::optional<at::Tensor> maybe_top_k_arr,
                        unsigned int top_k_val, int64_t hip_stream);

// top p renorm probs
void top_p_renorm_probs(at::Tensor probs, at::Tensor renorm_probs, std::optional<at::Tensor> maybe_top_p_arr,
                        double top_p_val, int64_t hip_stream);

// top k top p sampling from probs
void top_k_top_p_sampling_from_probs(at::Tensor probs, at::Tensor uniform_samples, at::Tensor samples,
                                     at::Tensor success, std::optional<at::Tensor> maybe_top_k_arr, double top_k_val,
                                     std::optional<at::Tensor> maybe_top_p_arr, double top_p_val, bool deterministic,
                                     int64_t hip_stream);

// top p sampling from probs
void top_p_sampling_from_probs(at::Tensor probs, at::Tensor uniform_samples, at::Tensor samples, at::Tensor success,
                               std::optional<at::Tensor> maybe_top_p_arr, double top_p_val, bool deterministic,
                               int64_t hip_stream);

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  // trt_reduce
  m.def("init_custom_ar", &init_custom_ar, "init custom allreduce meta (CUDA)");
  m.def("dispose", &dispose, "dispose custom allreduce meta");
  m.def("all_reduce", &all_reduce, "custom all reduce (CUDA)");
  m.def("get_graph_buffer_ipc_meta", &get_graph_buffer_ipc_meta, "custom all reduce get graph ipc meta");
  m.def("register_graph_buffers", &register_graph_buffers, "custom all reduce register graph buffers");
  // moe_align_block_size
  m.def("moe_align_block_size", &moe_align_block_size, "MOE Align Block Size (CUDA)");
  // sampling_scaling_penalties
  m.def("sampling_scaling_penalties", &sampling_scaling_penalties, "Sampling scaling penalties (CUDA)");
  // int8_scaled_mm
  m.def("int8_scaled_mm", &int8_scaled_mm, "INT8 scaled matmul (CUDA)");
  // lightning_attention_decode
  m.def("lightning_attention_decode", &lightning_attention_decode, "Lightning Attention Ddecode (CUDA)");
  // rotary embedding
  m.def("rotary_embedding", &rotary_embedding, "Rotary Embedding (CUDA)");
  // rms norm
  m.def("rmsnorm", &rmsnorm, "RMSNorm (CUDA)");
  // fused rms norm
  m.def("fused_add_rmsnorm", &fused_add_rmsnorm, "Fused Add RMSNorm (CUDA)");
  // gemma rms norm
  m.def("gemma_rmsnorm", &gemma_rmsnorm, "Gemma RMSNorm (CUDA)");
  // fused gemma rms norm
  m.def("gemma_fused_add_rmsnorm", &gemma_fused_add_rmsnorm, "Gemma Fused Add RMSNorm (CUDA)");
  // silu and mul
  m.def("silu_and_mul", &silu_and_mul, "Silu and Mul (CUDA)");
  // gelu tanh and mul
  m.def("gelu_tanh_and_mul", &gelu_tanh_and_mul, "Gelu Tanh and Mul (CUDA)");
  // gelu and mul
  m.def("gelu_and_mul", &gelu_and_mul, "Gelu and Mul (CUDA)");
  // bmm fp8
  m.def("bmm_fp8", &bmm_fp8, "BMM FP8 (CUDA)");
  // min p sampling from probs
  m.def("min_p_sampling_from_probs", &min_p_sampling_from_probs, "Min P Sampling From Probs (CUDA)");
  // top k renorm probs
  m.def("top_k_renorm_probs", &top_k_renorm_probs, "Top K Renorm Probs (CUDA)");
  // top p renorm probs
  m.def("top_p_renorm_probs", &top_p_renorm_probs, "Top P Renorm Probs (CUDA)");
  // top k top p sampling from probs
  m.def("top_k_top_p_sampling_from_probs", &top_k_top_p_sampling_from_probs, "Top K Top P Sampling From Probs (CUDA)");
  // top p sampling from probs
  m.def("top_p_sampling_from_probs", &top_p_sampling_from_probs, "Top P Sampling From Probs (CUDA)");
}
